#include "hip/hip_runtime.h"
#include "gpu/cost_functions/binary_function.h"
#include "gpu/cost_functions/cost_function_kernel.h"
#include "gpu/cost_functions/unary_function.h"
#include "gpu/gpu_displacement_field.h"
#include "hybrid_graph_cut_optimizer.h"

#include <stk/cuda/hip/hip_runtime.h>
#include <stk/cuda/stream.h>
#include <stk/cuda/volume.h>

namespace cuda {
    using namespace stk::cuda;
}

template<typename UpdateFn>
__global__ void apply_displacement_delta_kernel(
    cuda::VolumePtr<float4> df_in,
    dim3 dims,
    cuda::VolumePtr<uint8_t> labels,
    float4 delta,
    float3 inv_spacing,
    cuda::VolumePtr<float4> df_out
)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= dims.x ||
        y >= dims.y ||
        z >= dims.z)
    {
        return;
    }

    if (labels(x,y,z)) {
        UpdateFn fn;
        float3 d = fn(df_in, dims, inv_spacing, int3{x,y,z}, delta);
        df_out(x,y,z) = {d.x, d.y, d.z};
    }
}

void apply_displacement_delta(
    GpuDisplacementField& df_in,
    GpuDisplacementField& df_out,
    stk::GpuVolume& labels,
    const float3& delta,
    Settings::UpdateRule update_rule,
    cuda::Stream stream
)
{
    dim3 dims = df_in.size();
    dim3 block_size {32,32,1};
    dim3 grid_size {
        (dims.x + block_size.x - 1) / block_size.x,
        (dims.y + block_size.y - 1) / block_size.y,
        (dims.z + block_size.z - 1) / block_size.z
    };

    float3 inv_spacing {
        1.0f / df_in.spacing().x,
        1.0f / df_in.spacing().y,
        1.0f / df_in.spacing().z
    };

    if (update_rule == Settings::UpdateRule_Additive) {
        // In and out buffer for displacement field in the additive case can 
        //  be the same, since all updates are guaranteed to be independent.
        apply_displacement_delta_kernel<cuda::AdditiveUpdate>
        <<<grid_size, block_size, 0, stream>>>(
            df_out.volume(),
            dims,
            labels,
            float4{delta.x, delta.y, delta.z, 0.0f},
            inv_spacing,
            df_out.volume()
        );
    }
    else if (update_rule == Settings::UpdateRule_Compositive) {
        apply_displacement_delta_kernel<cuda::CompositiveUpdate>
        <<<grid_size, block_size, 0, stream>>>(
            df_in.volume(),
            dims,
            labels,
            float4{delta.x, delta.y, delta.z, 0.0f},
            inv_spacing,
            df_out.volume()
        );
    }
    CUDA_CHECK_ERRORS(hipPeekAtLastError());
}

__global__ void reduce_total_energy(
    cuda::VolumePtr<float2> unary_term,
    cuda::VolumePtr<float4> binary_term_x, // Regularization cost in x+
    cuda::VolumePtr<float4> binary_term_y, // y+
    cuda::VolumePtr<float4> binary_term_z, // z+
    dim3 dims,
    float* out
)
{
    extern __shared__ float shared[];

    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int bid = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;

    shared[tid] = 0;

    if (x < dims.x &&
        y < dims.y &&
        z < dims.z)
    {
        float e = unary_term(x,y,z).x;
        if (x + 1 < int(dims.x)) {
            e += binary_term_x(x,y,z).x;
        }
        if (y + 1 < int(dims.y)) {
            e += binary_term_y(x,y,z).x;
        }
        if (z + 1 < int(dims.z)) {
            e += binary_term_z(x,y,z).x;
        }

        shared[tid] = e;
    }
    __syncthreads();

    #define REDUCTION_STEP(n_) \
        if (tid < (n_)) shared[tid] = shared[tid] + shared[tid+(n_)]; \
        __syncthreads();

    REDUCTION_STEP(512);
    REDUCTION_STEP(256);
    REDUCTION_STEP(128);
    REDUCTION_STEP(64);
    REDUCTION_STEP(32);
    REDUCTION_STEP(16);
    REDUCTION_STEP(8);
    REDUCTION_STEP(4);
    REDUCTION_STEP(2);

    if (tid == 0) {
        out[bid] = shared[0] + shared[1];
    }

    #undef REDUCTION_STEP
}

double calculate_energy(
    GpuUnaryFunction& unary_fn,
    GpuBinaryFunction& binary_fn,
    GpuDisplacementField& df,
    stk::GpuVolume& unary_cost,
    stk::GpuVolume& binary_cost_x,
    stk::GpuVolume& binary_cost_y,
    stk::GpuVolume& binary_cost_z
)
{
    // Reset unary cost
    hipExtent extent = make_hipExtent(
        unary_cost.size().x * sizeof(float2),
        unary_cost.size().y,
        unary_cost.size().z
    );
    CUDA_CHECK_ERRORS(hipMemset3D(unary_cost.pitched_ptr(), 0, extent));

    dim3 dims = unary_cost.size();
    int3 begin {0, 0, 0};
    int3 end {(int)dims.x, (int)dims.y, (int)dims.z};

    cuda::Stream& stream = stk::cuda::Stream::null();

    // Update rule doesn't matter in this case since we don't want the energy for a move.
    unary_fn(df, {0,0,0}, begin, end, Settings::UpdateRule_Additive, unary_cost, stream);

    // Compute binary terms
    binary_fn(
        df,
        {0, 0, 0},
        begin,
        end,
        Settings::UpdateRule_Additive,
        binary_cost_x,
        binary_cost_y,
        binary_cost_z,
        stream
    );

    dim3 block_size{32,32,1};

    dim3 grid_size {
        (dims.x + block_size.x - 1) / block_size.x,
        (dims.y + block_size.y - 1) / block_size.y,
        (dims.z + block_size.z - 1) / block_size.z
    };
    uint32_t n_blocks = grid_size.x * grid_size.y * grid_size.z;

    float* d_block_sum;
    CUDA_CHECK_ERRORS(hipMalloc(&d_block_sum, n_blocks*sizeof(float)));

    reduce_total_energy<<<grid_size, block_size,
        uint32_t(sizeof(float)*1024)>>>
    (
        unary_cost,
        binary_cost_x,
        binary_cost_y,
        binary_cost_z,
        dims,
        d_block_sum
    );

    CUDA_CHECK_ERRORS(hipPeekAtLastError());
    CUDA_CHECK_ERRORS(hipDeviceSynchronize());

    float* block_sum = new float[n_blocks];
    CUDA_CHECK_ERRORS(hipMemcpy(block_sum, d_block_sum, n_blocks*sizeof(float), hipMemcpyDeviceToHost));

    // TODO: Perform all reduction on GPU
    double total_energy = 0;
    for (int i = 0; i < (int)n_blocks; ++i) {
        total_energy += block_sum[i];
    }

    delete [] block_sum;
    CUDA_CHECK_ERRORS(hipFree(d_block_sum));

    return total_energy;
}
