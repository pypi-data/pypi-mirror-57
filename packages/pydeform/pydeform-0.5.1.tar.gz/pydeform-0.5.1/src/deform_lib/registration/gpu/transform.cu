#include "hip/hip_runtime.h"
#include "gpu_displacement_field.h"
#include "transform.h"

#include <stk/common/assert.h>
#include <stk/cuda/hip/hip_runtime.h>
#include <stk/cuda/volume.h>
#include <stk/image/gpu_volume.h>
#include <stk/math/float3.h>
#include <stk/math/float4.h>

namespace cuda {
    using namespace stk::cuda;
}

template<typename T>
__global__ void transform_kernel_linear(
    cuda::VolumePtr<T> src,
    dim3 src_dims,
    cuda::DisplacementField<> df, // We don't care about update rule since we don't update
    float3 moving_origin,
    float3 inv_moving_spacing,
    Matrix3x3f inv_moving_direction,
    cuda::VolumePtr<T> out
)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= df.size().x ||
        y >= df.size().y ||
        z >= df.size().z)
    {
        return;
    }

    float3 mp = inv_moving_direction * (df.transform_index(int3{x,y,z}) - moving_origin)
                * inv_moving_spacing;

    out(x,y,z) = cuda::linear_at_border(src, src_dims, mp.x, mp.y, mp.z);
}

template<typename T>
__global__ void transform_kernel_nn(
    cuda::VolumePtr<T> src,
    dim3 src_dims,
    cuda::DisplacementField<> df, // We don't care about update rule since we don't update
    float3 moving_origin,
    float3 inv_moving_spacing,
    Matrix3x3f inv_moving_direction,
    cuda::VolumePtr<T> out
)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if (x >= df.size().x ||
        y >= df.size().y ||
        z >= df.size().z)
    {
        return;
    }

    float3 mp = inv_moving_direction * (df.transform_index(int3{x,y,z}) - moving_origin) 
                * inv_moving_spacing;

    int xt = roundf(mp.x);
    int yt = roundf(mp.y);
    int zt = roundf(mp.z);

    if (xt >= 0 && xt < src_dims.x &&
        yt >= 0 && yt < src_dims.y &&
        zt >= 0 && zt < src_dims.z) {

        out(x,y,z) = src(xt, yt, zt);
    }
    else {
        out(x,y,z) = T{0};
    }
}

static void run_nn_kernel(
    stk::Type type,
    const dim3& grid_size,
    const dim3& block_size,
    const stk::GpuVolume& src,
    const GpuDisplacementField& df,
    stk::GpuVolume& out
)
{
    float3 inv_moving_spacing = float3{1.0f, 1.0f, 1.0f} / src.spacing();

    #define TRANSFORM_KERNEL_NN(type) \
            transform_kernel_nn<type><<<grid_size, block_size>>>( \
                src, \
                src.size(), \
                cuda::DisplacementField<>(df), \
                src.origin(), \
                inv_moving_spacing, \
                src.inverse_direction(), \
                out \
            )

    switch (type) {
    case stk::Type_Char:  TRANSFORM_KERNEL_NN(char);  break;
    case stk::Type_Char2: TRANSFORM_KERNEL_NN(char2); break;
    case stk::Type_Char4: TRANSFORM_KERNEL_NN(char4); break;

    case stk::Type_UChar:  TRANSFORM_KERNEL_NN(uint8_t); break;
    case stk::Type_UChar2: TRANSFORM_KERNEL_NN(uchar2);  break;
    case stk::Type_UChar4: TRANSFORM_KERNEL_NN(uchar4);  break;

    case stk::Type_Short:  TRANSFORM_KERNEL_NN(short);  break;
    case stk::Type_Short2: TRANSFORM_KERNEL_NN(short2); break;
    case stk::Type_Short4: TRANSFORM_KERNEL_NN(short4); break;

    case stk::Type_UShort:  TRANSFORM_KERNEL_NN(uint16_t); break;
    case stk::Type_UShort2: TRANSFORM_KERNEL_NN(ushort2);  break;
    case stk::Type_UShort4: TRANSFORM_KERNEL_NN(ushort4);  break;

    case stk::Type_Int:  TRANSFORM_KERNEL_NN(int);  break;
    case stk::Type_Int2: TRANSFORM_KERNEL_NN(int2); break;
    case stk::Type_Int4: TRANSFORM_KERNEL_NN(int4); break;

    case stk::Type_UInt:  TRANSFORM_KERNEL_NN(uint32_t); break;
    case stk::Type_UInt2: TRANSFORM_KERNEL_NN(uint2);    break;
    case stk::Type_UInt4: TRANSFORM_KERNEL_NN(uint4);    break;

    case stk::Type_Float:  TRANSFORM_KERNEL_NN(float);  break;
    case stk::Type_Float2: TRANSFORM_KERNEL_NN(float2); break;
    case stk::Type_Float4: TRANSFORM_KERNEL_NN(float4); break;

    default:
        FATAL() << "Unsupported pixel type";
    };

    #undef TRANSFORM_KERNEL_NN
}

static void run_linear_kernel(
    stk::Type type,
    const dim3& grid_size,
    const dim3& block_size,
    const stk::GpuVolume& src,
    const GpuDisplacementField& df,
    stk::GpuVolume& out
)
{
    #define TRANSFORM_KERNEL_LINEAR(type) \
            transform_kernel_linear<type><<<grid_size, block_size>>>( \
                src, \
                src.size(), \
                df, \
                src.origin(), \
                inv_moving_spacing, \
                src.inverse_direction(), \
                out \
            )

    float3 inv_moving_spacing = float3{1.0f, 1.0f, 1.0f} / src.spacing();

    switch (type) {

    case stk::Type_Float:  TRANSFORM_KERNEL_LINEAR(float);  break;
    case stk::Type_Float2: TRANSFORM_KERNEL_LINEAR(float2); break;
    case stk::Type_Float4: TRANSFORM_KERNEL_LINEAR(float4); break;

    default:
        FATAL() << "Interpolation mode only supports float types";
    };

    #undef TRANSFORM_KERNEL_LINEAR
}


stk::GpuVolume gpu::transform_volume(
    const stk::GpuVolume& src,
    const GpuDisplacementField& df,
    transform::Interp i,
    const dim3& block_size
)
{
    ASSERT(src.usage() == stk::gpu::Usage_PitchedPointer);

    dim3 dims = df.size();

    stk::GpuVolume out(dims, src.voxel_type());
    out.copy_meta_from(df.volume());

    dim3 grid_size {
        (dims.x + block_size.x - 1) / block_size.x,
        (dims.y + block_size.y - 1) / block_size.y,
        (dims.z + block_size.z - 1) / block_size.z
    };

    if (i == transform::Interp_NN) {
        run_nn_kernel(src.voxel_type(), grid_size, block_size, src, df, out);
    } else {
        run_linear_kernel(src.voxel_type(), grid_size, block_size, src, df, out);
    }

    CUDA_CHECK_ERRORS(hipPeekAtLastError());
    CUDA_CHECK_ERRORS(hipDeviceSynchronize());

    return out;
}
